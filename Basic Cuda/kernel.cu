#include <iostream>
#include "hip/hip_runtime.h"




using namespace std;

__global__ void cube(long *deviceOutput, long *deviceInput)
{
	int idx = threadIdx.x;
	long f = deviceInput[idx];
	deviceOutput[idx] = f * f * f;
}

int main()
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	
	const int ArraySize = 1024;
	const int ArrayBytes = ArraySize * sizeof(long);
	
	//ArrayBytes = 4 * ArraySize.  Because the dIn and dOut are both floating 
	//Numbers, meaning I need to know the size to allocate when doing Malloc
	//And Memcpy.  
	//cout << ArrayBytes << endl;

	

	long hostInput[ArraySize];
	
	for (int i = 0; i < ArraySize; i++) {
		hostInput[i] = i;
	}

	long hostOutput[ArraySize];

	//GPU memory pointers
	long * deviceInput;
	long * deviceOutput;

	

	//Allocate GPU memory
	hipMalloc((void **)&deviceInput, ArrayBytes);
	hipMalloc((void **)&deviceOutput, ArrayBytes);

	//Transfer the array to GPU
	hipMemcpy(deviceInput, hostInput, ArrayBytes, hipMemcpyHostToDevice);

	//Launch the Kernal
	//This Kernal has 1 Thread Block, and that thread block has ArraySize amount of Threads.
	hipEventRecord(start);
	cube<<<2, ArraySize >>>(deviceOutput, deviceInput);
	hipEventRecord(stop);
	//Copy back result from GPU to CPU

	hipMemcpy(hostOutput, deviceOutput, ArrayBytes, hipMemcpyDeviceToHost);

	for (int i = 0; i < ArraySize; i++) {
		cout << hostOutput[i] << endl;
	}
	

	hipFree(deviceInput);
	hipFree(deviceOutput);
	float ms = 0;
	hipEventElapsedTime(&ms, start, stop);
	cout << "Milliseconds: " << ms << endl;

	hipEventDestroy(start);
	hipEventDestroy(stop);
	//End of program.
	
	return 0;
}
